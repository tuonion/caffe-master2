#include <vector>

#include "caffe/layers/base_data_layer.hpp"

namespace caffe {

template<typename Dtype>
void BasePrefetchingDataLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
	Batch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");

//  LOG(INFO)<<"data_string:"<<batch->data_.toString();
//  LOG(INFO)<<"label_string:"<<batch->label_.toString();
	// Reshape to loaded data.
	top[0]->ReshapeLike(batch->data_);

//	const Dtype* cpu_label = batch->label_.cpu_data();
//	ostringstream osm;
//	for (int i = 0; i < 8; ++i) {
//		for (int j = 0; j < 15; j++) {
//			osm << cpu_label[i * 15 + j] << " ";
//		}
//		osm << "\n";
//	}
//	LOG_IF(INFO,gb_logDataFlag) << "label_data:\n" << osm.str();


	// Copy the data
	caffe_copy(batch->data_.count(), batch->data_.gpu_data(), top[0]->mutable_gpu_data());
	if (this->output_labels_) {
		// Reshape to loaded labels.
		top[1]->ReshapeLike(batch->label_);
		// Copy the labels.
		caffe_copy(batch->label_.count(), batch->label_.gpu_data(), top[1]->mutable_gpu_data());
	}
	// Ensure the copy is synchronous wrt the host, so that the next batch isn't
	// copied in meanwhile.
	CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
	prefetch_free_.push(batch);
}

INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);

}  // namespace caffe
